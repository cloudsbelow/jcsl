#include <iostream>
#include <map>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>

#include "globals.cuh"

int CUDA_SAFE_CALL(hipError_t err){
  if (hipSuccess != err) {                                         
    const char *errStr;
    hipDrvGetErrorString(err, &errStr);
    fprintf(stderr, "CUDA Error: %s (error code: %d)\n", errStr, err);
  } 
  return err;
} 
int CUDA_SAFE_CALL(hipError_t err){
  if(hipSuccess != err) {
    fprintf(stderr, "CUDA Error_t:  %d\n", err);
  }
  return err;
}


namespace Globals {
  void donothing(hipFunction_t func) {
      // Who woulda guessed
  }

  // Definitions of global objects
  Collection<void*, hipError_t, &hipFree> devptrs;
  Collection<hipModule_t, hipError_t, &hipModuleUnload> modules;
  Collection<hipFunction_t, void, donothing> functions;
}